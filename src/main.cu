#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstddef>
#include <iostream>
#include <numeric>
#include <map>
#include <time.h>
#include <valarray>
#include <string>

#include <hdf5.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "range.hpp"
#include "utils.hpp"

#define NUM_ROWS 28
#define NUM_COLS 28
#define NUM_CHANNELS 1
#define NUM_DIGITS 10

#define TILE_WIDTH 16
#define MAX_THREADS 1024

static int FLAGS_batch_size = 10000;
static std::string FLAGS_testdata{};
static std::string FLAGS_model{};

// Data and reference data dimensions
static int xdims[] = { FLAGS_batch_size, NUM_ROWS, NUM_COLS, NUM_CHANNELS };
static int rdims[] = { FLAGS_batch_size, NUM_DIGITS };

// Model dimensions
static int conv1dims[] = { 5, 5, 1, 32 };
static int conv2dims[] = { 5, 5, 32, 64 };
static int fc1dims[] = { 1024, 128 };
static int fc2dims[] = { 128, 10 };

struct dims {
	int dim[4];
};

// __global__ void unroll_x_kernel(float *X, float *X_unroll, dims x, dims w, dims y) {
// 	int c, s, h_out, w_out, h_unroll, w_unroll, xoffset, offset;
// 	int index = blockDim.x * blockIdx.x + threadIdx.x;
// 	int H_filter = w.dim[0];
// 	int W_filter = w.dim[1];
// 	int H_out = y.dim[1];
// 	int W_out = y.dim[2];
// 	int C = w.dim[2];
// 	int W_unroll = H_out * W_out;
//
// 	if (index < C * W_unroll) {
// 		c = index / W_unroll;
// 		s = index % W_unroll;
// 		h_out = s / W_out;
// 		w_out = s % W_out;
// 		for (int p = 0; p < H_filter; p++) {
// 			for (int q = 0; q < W_filter; q++) {
// 				w_unroll = s;
// 				h_unroll = c * H_filter * W_filter + p * W_filter + q;
//         offset = W_unroll * h_unroll + w_unroll;
// 				xoffset = ((h_out + p) * x.dim[2] + (w_out + q)) * x.dim[3] + c;
// 				X_unroll[offset] = X[xoffset];
// 			}
// 		}
// 	}
// }

__global__ void unroll_x_kernel(float *X, float *X_unroll, dims x, dims w, dims y) {
	int w_unroll, h_unroll, uoffset, xoffset;
	int H_filter = w.dim[0];
	int W_filter = w.dim[1];
	int H_out = y.dim[1];
	int W_out = y.dim[2];
	int W_unroll = H_out * W_out;
	int h_out = blockIdx.x;
	int w_out = blockIdx.y;
	int c = threadIdx.x;
	int p = threadIdx.y;
	int q = threadIdx.z;

	if (h_out < y.dim[1] && w_out < y.dim[2] && p < w.dim[0] && q < w.dim[1] && c < w.dim[2]) {
		w_unroll = h_out * W_out + w_out;
		h_unroll = c * H_filter * W_filter + p * W_filter + q;
		uoffset = h_unroll * W_unroll + w_unroll;
		xoffset = ((h_out + p) * x.dim[2] + (w_out + q)) * x.dim[3] + c;
		X_unroll[uoffset] = X[xoffset];
	}
}

__global__ void reroll_y_kernel(float *Y, float *Y_roll, dims y) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int y_roll_row = index / (y.dim[1] * y.dim[2]);
	int y_roll_col = index % (y.dim[1] * y.dim[2]);
	int y_row = y_roll_col / y.dim[2];
	int y_col = y_roll_col % y.dim[2];

	if (index < y.dim[1] * y.dim[2] * y.dim[3]) {
		int yroll_offset = y_row * y.dim[2] * y.dim[3] + y_col * y.dim[3] + y_roll_row;
		int y_offset = y_roll_row * y.dim[1] * y.dim[2] + y_roll_col;
		Y_roll[yroll_offset] = Y[y_offset];
	}
}

__global__ void matrixMultiplyShared(float *A, float *B, float *C,
	int numARows, int numAColumns,
	int numBRows, int numBColumns,
	int numCRows, int numCColumns) {

	__shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
	__shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Row = by * blockDim.y + ty;
	int Col = bx * blockDim.x + tx;
	float Cvalue = 0.0;
	int numOfTiles = numAColumns / TILE_WIDTH;
	if (numAColumns % TILE_WIDTH) numOfTiles++;

	for (int m = 0; m < numOfTiles; m++) {
		if ((m * TILE_WIDTH + tx < numAColumns) && (Row < numARows)) {
			subTileA[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
		}
		else {
			subTileA[ty][tx] = 0.0;
		}
		if ((m * TILE_WIDTH + ty < numBRows) && (Col < numBColumns)) {
			subTileB[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
		}
		else {
			subTileB[ty][tx] = 0.0;
		}
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; k++) {
			Cvalue += subTileA[ty][k] * subTileB[k][tx];
		}
		__syncthreads();
	}

	if (Row < numCRows && Col < numCColumns) {
		C[Row * numBColumns + Col] = (Cvalue < 0) ? 0 : Cvalue;
	}
}

__global__ void average_pool_kernel(float *X, float *Y, int pool_size, dims x, dims y) {
	int xoffset, yoffset;
	int n = blockIdx.x;
	int m = blockIdx.y;
	int h = threadIdx.x;
	int w = threadIdx.y;
	float acc = 0;

	for (int p = 0; p < pool_size; p++) {
		for (int q = 0; q < pool_size; q++) {
      if (n < y.dim[0] && m < y.dim[3] && w < y.dim[2] && h < y.dim[1]) {
        xoffset = ((n * x.dim[1] + (pool_size * h + p)) * x.dim[2] + (pool_size * w + q)) * x.dim[3] + m;
  			acc += X[xoffset] / (1.0f * pool_size * pool_size);
      }
		}
	}

  if (n < y.dim[0] && m < y.dim[3] && w < y.dim[2] && h < y.dim[1]) {
    yoffset = ((n * y.dim[1] + h) * y.dim[2] + w) * y.dim[3] + m;
  	Y[yoffset] = acc;
  }
}

void average_pool_parallel(const float *x, float *y, const int xdims[4], const int ydims[4], const int pool_size) {
	float *device_x, *device_y;

	dims y_d, x_d;
	for (int i = 0; i < 4; i++) {
		y_d.dim[i] = ydims[i];
		x_d.dim[i] = xdims[i];
	}

	int size_x = sizeof(float) * xdims[0] * xdims[1] * xdims[2] * xdims[3];
	int size_y = sizeof(float) * ydims[0] * ydims[1] * ydims[2] * ydims[3];

	hipMalloc((void **)&device_x, size_x);
	hipMalloc((void **)&device_y, size_y);

	hipMemcpy(device_x, x, size_x, hipMemcpyHostToDevice);

  hipDeviceSynchronize();

	dim3 DimGrid(ydims[0], ydims[3], 1);
	dim3 DimBlock(ydims[1], ydims[2], 1);

	average_pool_kernel <<<DimGrid, DimBlock>>> (device_x, device_y, pool_size, x_d, y_d);

  hipDeviceSynchronize();

	hipMemcpy(y, device_y, size_y, hipMemcpyDeviceToHost);

	hipFree(device_x);
	hipFree(device_y);
}

void unroll_weights(const float *W, float *W_unroll, dims w) {
	int c, m, row, col;
	int unroll_offset, offset;
	int filter_h = w.dim[0];
	int filter_w = w.dim[1];
	int C = w.dim[2];
	int M = w.dim[3];
	for (row = 0; row < filter_h; row++) {
		for (col = 0; col < filter_w; col++) {
			for (c = 0; c < C; c++) {
				for (m = 0; m < M; m++) {
					unroll_offset = ((m * C + c) * filter_h + row) * filter_w + col;
					offset = ((row * filter_w + col) * C + c) * M + m;
					W_unroll[unroll_offset] = W[offset];
				}
			}
		}
	}
}
void conv_forward_unroll(const float *x, const float *w, float *y, const int xdims[4], const int wdims[4], const int ydims[4]) {
	float *device_x, *device_y, *device_x_unroll, *device_w_unroll, *device_y_unroll;

	// std::cout<< "\nINPUT DIMENSIONS:\n";
	// std::cout<< "N: "<< xdims[0] << ", H: "<< xdims[1] << ", W: "<< xdims[2] << ", C: "<< xdims[3] << "\n";
	// std::cout<< "K1: "<< wdims[0] << ", K2: "<< wdims[1] << ", C: "<< wdims[2] << ", M: "<< wdims[3] << "\n";
	// std::cout<< "N: "<< ydims[0] << ", H_Out: "<< ydims[1] << ", W_Out: "<< ydims[2] << ", M: "<< ydims[3] << "\n\n";

	hipStream_t stream0, stream1,stream2,stream3,stream4;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);
	hipStreamCreate(&stream4);

	dims y_d, w_d, x_d;
	for (int i = 0; i < 4; i++) {
		y_d.dim[i] = ydims[i];
		x_d.dim[i] = xdims[i];
		w_d.dim[i] = wdims[i];
	}

	int numAColumns = wdims[0] * wdims[1] * wdims[2], numARows = ydims[3];
	int numBColumns = ydims[1] * ydims[2], numBRows = wdims[0] * wdims[1] * wdims[2];
	int numCColumns = numBColumns, numCRows = numARows;

	int size_x = sizeof(float) * xdims[0] * xdims[1] * xdims[2] * xdims[3];
	int size_y = sizeof(float) * ydims[0] * ydims[1] * ydims[2] * ydims[3];
	int size_x_unroll = sizeof(float) * wdims[0] * wdims[1] * wdims[2] * ydims[1] * ydims[2];
	int size_w_unroll = sizeof(float) * wdims[0] * wdims[1] * wdims[2] * ydims[3];
	int size_y_unroll = sizeof(float) * ydims[1] * ydims[2] * ydims[3];

	int stripe_x = xdims[1] * xdims[2] * xdims[3];
	int stripe_y = ydims[1] * ydims[2] * ydims[3];
	int stripe_x_unroll = wdims[0] * wdims[1] * wdims[2] * ydims[1] * ydims[2];
	int stripe_y_unroll = ydims[1] * ydims[2] * ydims[3];

	hipMalloc((void **)&device_x, size_x);
	hipMalloc((void **)&device_y, size_y);
	hipMalloc((void **)&device_x_unroll, size_x_unroll* xdims[0]);
	hipMalloc((void **)&device_w_unroll, size_w_unroll);
	hipMalloc((void **)&device_y_unroll, size_y_unroll* xdims[0]);

	float * w_unroll = (float *)malloc(size_w_unroll * sizeof(float));
	unroll_weights(w, w_unroll, w_d);

	
	
	hipMemcpy(device_w_unroll, w_unroll, size_w_unroll, hipMemcpyHostToDevice);

	// dim3 DimBlock_unroll_x(MAX_THREADS, 1, 1);
	// dim3 DimGrid_unroll_x(ceil((float)(wdims[2] * ydims[1] * ydims[2]) / MAX_THREADS), 1, 1);

	dim3 DimBlock_unroll_x(wdims[2], wdims[0], wdims[1]);
	dim3 DimGrid_unroll_x(ydims[1], ydims[2], 1);

	dim3 DimBlock_matmul(TILE_WIDTH, TILE_WIDTH, 1);
	dim3 DimGrid_matmul(ceil((float)(ydims[1] * ydims[2]) / TILE_WIDTH), ceil((float)(ydims[3]) / TILE_WIDTH), 1);

	dim3 DimBlock_reroll_y(MAX_THREADS, 1, 1);
	dim3 DimGrid_reroll_y(ceil((float)(ydims[1] * ydims[2] * ydims[3]) / MAX_THREADS), 1, 1);
	
	hipMemcpyAsync(device_x, x, stripe_x *sizeof(float), hipMemcpyHostToDevice,stream0);
	hipDeviceSynchronize();
	hipMemcpyAsync(device_x + stripe_x, x + stripe_x, stripe_x *sizeof(float), hipMemcpyHostToDevice,stream1);
	unroll_x_kernel <<<DimGrid_unroll_x, DimBlock_unroll_x,0,stream0>>> (device_x , device_x_unroll, x_d, w_d, y_d);
	hipDeviceSynchronize();
	hipMemcpyAsync(device_x + stripe_x *2, x + 2* stripe_x, stripe_x *sizeof(float), hipMemcpyHostToDevice,stream2);
	unroll_x_kernel <<<DimGrid_unroll_x, DimBlock_unroll_x,0,stream1>>> (device_x +  stripe_x, device_x_unroll+ stripe_x_unroll, x_d, w_d, y_d);
	matrixMultiplyShared <<<DimGrid_matmul, DimBlock_matmul,0,stream0>>> (device_w_unroll, device_x_unroll, device_y_unroll,numARows, numAColumns,numBRows, numBColumns,numCRows, numCColumns);
	hipDeviceSynchronize();
	hipMemcpyAsync(device_x + stripe_x *3, x + 3* stripe_x, stripe_x *sizeof(float), hipMemcpyHostToDevice,stream3);
	unroll_x_kernel <<<DimGrid_unroll_x, DimBlock_unroll_x,0,stream2>>> (device_x +  2 *stripe_x, device_x_unroll+ 2*stripe_x_unroll, x_d, w_d, y_d);
	matrixMultiplyShared <<<DimGrid_matmul, DimBlock_matmul,0,stream1>>> (device_w_unroll, device_x_unroll+stripe_x_unroll, device_y_unroll+stripe_y_unroll,numARows, numAColumns,numBRows, numBColumns,numCRows, numCColumns);
	reroll_y_kernel <<<DimGrid_reroll_y, DimBlock_reroll_y,0,stream0>>> (device_y_unroll , device_y , y_d);
	hipDeviceSynchronize();



	for (int i = 0; i < xdims[0]-4; i++) {
		hipMemcpyAsync(device_x+ (i+4) * stripe_x, x+ (i+4) * stripe_x, stripe_x *sizeof(float), hipMemcpyHostToDevice,stream4);		
		unroll_x_kernel <<<DimGrid_unroll_x, DimBlock_unroll_x,0,stream3>>> (device_x + (i +3)* stripe_x, device_x_unroll+(i +3)* stripe_x_unroll, x_d, w_d, y_d);
		matrixMultiplyShared <<<DimGrid_matmul, DimBlock_matmul,0,stream2>>> (device_w_unroll , device_x_unroll+ (i +2)* stripe_x_unroll, device_y_unroll+ (i +2)* stripe_y_unroll,numARows, numAColumns,numBRows, numBColumns,numCRows, numCColumns);
		reroll_y_kernel <<<DimGrid_reroll_y, DimBlock_reroll_y,0,stream1>>> (device_y_unroll + (i+1) * stripe_y_unroll, device_y + (i+1) * stripe_y, y_d);
		hipMemcpyAsync(y + i * stripe_y, device_y + i* stripe_y, stripe_y * sizeof(float), hipMemcpyDeviceToHost,stream0);
		hipDeviceSynchronize();
	}
	unroll_x_kernel <<<DimGrid_unroll_x, DimBlock_unroll_x,0,stream4>>> (device_x + (xdims[0] - 1)* stripe_x, device_x_unroll+(xdims[0] - 1)* stripe_x_unroll, x_d, w_d, y_d);
	matrixMultiplyShared <<<DimGrid_matmul, DimBlock_matmul,0,stream3>>> (device_w_unroll , device_x_unroll+ (xdims[0] - 2)* stripe_x_unroll, device_y_unroll+ (xdims[0] - 2)* stripe_y_unroll,numARows, numAColumns,numBRows, numBColumns,numCRows, numCColumns);
	reroll_y_kernel <<<DimGrid_reroll_y, DimBlock_reroll_y,0,stream2>>> (device_y_unroll + (xdims[0] - 3) * stripe_y_unroll, device_y + (xdims[0] - 3) * stripe_y, y_d);
	hipMemcpyAsync(y + (xdims[0] - 4) * stripe_y, device_y +(xdims[0] - 4)* stripe_y, stripe_y * sizeof(float), hipMemcpyDeviceToHost,stream1);
    hipDeviceSynchronize();
    matrixMultiplyShared <<<DimGrid_matmul, DimBlock_matmul,0,stream4>>> (device_w_unroll , device_x_unroll+ (xdims[0] - 1)* stripe_x_unroll, device_y_unroll+ (xdims[0] - 1)* stripe_y_unroll,numARows, numAColumns,numBRows, numBColumns,numCRows, numCColumns);
	reroll_y_kernel <<<DimGrid_reroll_y, DimBlock_reroll_y,0,stream3>>> (device_y_unroll + (xdims[0] - 2) * stripe_y_unroll, device_y + (xdims[0] - 2)* stripe_y, y_d);
	hipMemcpyAsync(y + (xdims[0] - 3) * stripe_y, device_y+(xdims[0] - 3) * stripe_y, stripe_y * sizeof(float), hipMemcpyDeviceToHost,stream2);
	hipDeviceSynchronize();
	reroll_y_kernel <<<DimGrid_reroll_y, DimBlock_reroll_y,0,stream4>>> (device_y_unroll + (xdims[0] - 1) * stripe_y_unroll, device_y + (xdims[0] - 1) * stripe_y, y_d);
	hipMemcpyAsync(y + (xdims[0] - 2) * stripe_y, device_y+(xdims[0] - 2) * stripe_y, stripe_y * sizeof(float), hipMemcpyDeviceToHost,stream3);
	hipDeviceSynchronize();
	hipMemcpyAsync(y + (xdims[0] - 1) * stripe_y, device_y +(xdims[0] - 1)* stripe_y, stripe_y * sizeof(float), hipMemcpyDeviceToHost,stream4);
	hipDeviceSynchronize();
	


	hipFree(device_x);
	hipFree(device_y);
	hipFree(device_y_unroll);
	hipFree(device_x_unroll);
	hipFree(device_w_unroll);
}

// void conv_forward_unroll(const float *x, const float *w, float *y, const int xdims[4], const int wdims[4], const int ydims[4]) {
// 	float *device_x, *device_y, *device_x_unroll, *device_w_unroll, *device_y_unroll;

// 	std::cout<< "\nINPUT DIMENSIONS:\n";
// 	std::cout<< "N: "<< xdims[0] << ", H: "<< xdims[1] << ", W: "<< xdims[2] << ", C: "<< xdims[3] << "\n";
// 	std::cout<< "K1: "<< wdims[0] << ", K2: "<< wdims[1] << ", C: "<< wdims[2] << ", M: "<< wdims[3] << "\n";
// 	std::cout<< "N: "<< ydims[0] << ", H_Out: "<< ydims[1] << ", W_Out: "<< ydims[2] << ", M: "<< ydims[3] << "\n\n";

// 	dims y_d, w_d, x_d;
// 	for (int i = 0; i < 4; i++) {
// 		y_d.dim[i] = ydims[i];
// 		x_d.dim[i] = xdims[i];
// 		w_d.dim[i] = wdims[i];
// 	}

// 	int numAColumns = wdims[0] * wdims[1] * wdims[2], numARows = ydims[3];
// 	int numBColumns = ydims[1] * ydims[2], numBRows = wdims[0] * wdims[1] * wdims[2];
// 	int numCColumns = numBColumns, numCRows = numARows;

// 	int size_x = sizeof(float) * xdims[0] * xdims[1] * xdims[2] * xdims[3];
// 	int size_y = sizeof(float) * ydims[0] * ydims[1] * ydims[2] * ydims[3];
// 	int size_x_unroll = sizeof(float) * wdims[0] * wdims[1] * wdims[2] * ydims[1] * ydims[2];
// 	int size_w_unroll = sizeof(float) * wdims[0] * wdims[1] * wdims[2] * ydims[3];
// 	int size_y_unroll = sizeof(float) * ydims[1] * ydims[2] * ydims[3];

// 	int stripe_x = xdims[1] * xdims[2] * xdims[3];
// 	int stripe_y = ydims[1] * ydims[2] * ydims[3];

// 	hipMalloc((void **)&device_x, size_x);
// 	hipMalloc((void **)&device_y, size_y);
// 	hipMalloc((void **)&device_x_unroll, size_x_unroll);
// 	hipMalloc((void **)&device_w_unroll, size_w_unroll);
// 	hipMalloc((void **)&device_y_unroll, size_y_unroll);

// 	float * w_unroll = (float *)malloc(size_w_unroll * sizeof(float));
// 	unroll_weights(w, w_unroll, w_d);

// 	hipMemcpy(device_x, x, size_x, hipMemcpyHostToDevice);
// 	hipMemcpy(device_w_unroll, w_unroll, size_w_unroll, hipMemcpyHostToDevice);

// 	// dim3 DimBlock_unroll_x(MAX_THREADS, 1, 1);
// 	// dim3 DimGrid_unroll_x(ceil((float)(wdims[2] * ydims[1] * ydims[2]) / MAX_THREADS), 1, 1);

// 	dim3 DimBlock_unroll_x(wdims[2], wdims[0], wdims[1]);
// 	dim3 DimGrid_unroll_x(ydims[1], ydims[2], 1);

// 	dim3 DimBlock_matmul(TILE_WIDTH, TILE_WIDTH, 1);
// 	dim3 DimGrid_matmul(ceil((float)(ydims[1] * ydims[2]) / TILE_WIDTH), ceil((float)(ydims[3]) / TILE_WIDTH), 1);

// 	dim3 DimBlock_reroll_y(MAX_THREADS, 1, 1);
// 	dim3 DimGrid_reroll_y(ceil((float)(ydims[1] * ydims[2] * ydims[3]) / MAX_THREADS), 1, 1);

// 	for (int i = 0; i < xdims[0]; i++) {
// 		unroll_x_kernel <<<DimGrid_unroll_x, DimBlock_unroll_x>>> (device_x + i * stripe_x, device_x_unroll, x_d, w_d, y_d);
// 		matrixMultiplyShared <<<DimGrid_matmul, DimBlock_matmul>>> (device_w_unroll, device_x_unroll, device_y_unroll,
// 			numARows, numAColumns,
// 			numBRows, numBColumns,
// 			numCRows, numCColumns);
// 		reroll_y_kernel <<<DimGrid_reroll_y, DimBlock_reroll_y>>> (device_y_unroll, device_y + i * stripe_y, y_d);
// 	}

// 	hipMemcpy(y, device_y, size_y, hipMemcpyDeviceToHost);

// 	hipFree(device_x);
// 	hipFree(device_y);
// 	hipFree(device_y_unroll);
// 	hipFree(device_x_unroll);
// 	hipFree(device_w_unroll);
// }

static int loadData(float *x, float *y) {
	// Open the data file
	const auto file_id =
		H5Fopen(FLAGS_testdata.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

	// Open the dataset x and y
	const auto x_id = H5Dopen2(file_id, "/x", H5P_DEFAULT);
	const auto y_id = H5Dopen2(file_id, "/y", H5P_DEFAULT);

	// Get the dataset x dimensions
	const auto xspace = H5Dget_space(x_id);
	const auto xndims = H5Sget_simple_extent_ndims(xspace);
	assert(xndims == 4);

	hsize_t *input_dims = allocate<hsize_t>(xndims);
	//hsize_t input_dims[xndims];
	H5Sget_simple_extent_dims(xspace, input_dims, NULL);
	if (input_dims[0] != FLAGS_batch_size) {
		std::cout << "data size does not match batch size specified!\n";
		return 1; // return error
	}
	std::cout << "input dimensions = " << input_dims[0] << " x " << input_dims[1]
		<< " x " << input_dims[2] << " x " << input_dims[3] << "\n";

	// Read the dataset x and y
	check_success(
		H5Dread(x_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, x));
	check_success(
		H5Dread(y_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, y));

	// Close the dataset x and y
	check_success(H5Dclose(x_id));
	check_success(H5Dclose(y_id));

	// Close the file
	check_success(H5Fclose(file_id));

	delete[] input_dims;

	// return success
	return 0;
}

static void loadModel(float *conv1, float *conv2, float *fc1, float *fc2) {
	// Open the model file
	const auto file_id = H5Fopen(FLAGS_model.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);

	// Open the dataset
	const auto conv1_id = H5Dopen2(file_id, "/conv1", H5P_DEFAULT);
	const auto conv2_id = H5Dopen2(file_id, "/conv2", H5P_DEFAULT);
	const auto fc1_id = H5Dopen2(file_id, "/fc1", H5P_DEFAULT);
	const auto fc2_id = H5Dopen2(file_id, "/fc2", H5P_DEFAULT);

	// Read the dataset
	check_success(H5Dread(conv1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
		H5P_DEFAULT, conv1));
	check_success(H5Dread(conv2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL,
		H5P_DEFAULT, conv2));
	check_success(
		H5Dread(fc1_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc1));
	check_success(
		H5Dread(fc2_id, H5T_NATIVE_FLOAT, H5S_ALL, H5S_ALL, H5P_DEFAULT, fc2));

	// Close the dataset x and y
	check_success(H5Dclose(conv1_id));
	check_success(H5Dclose(conv2_id));
	check_success(H5Dclose(fc1_id));
	check_success(H5Dclose(fc2_id));

	// Close the file
	check_success(H5Fclose(file_id));
}

// Recified linear unit 2d
static void relu2(float *X, const int xdims[2]) {
  for (const auto i : range(0, xdims[0] * xdims[1])) {
    X[i] = (X[i] < 0) ? 0 : X[i];
  }
}

__global__ void fully_forward_kernel(float *X, float *W, float *Y, int xdim0, int xdim1, int wdim1) {
  int i = blockIdx.x;
  int j = threadIdx.x;
  float sum = 0;

  for(int k = 0; k < xdim1; k++) {
    if (i < xdim0 && j < wdim1) {
      sum += X[i * xdim1 + k] * W[k * wdim1 + j];
    }
  }

  if (i < xdim0 && j < wdim1) {
    Y[i * wdim1 + j] = sum;
  }
}

void fully_forward_parallel(float *x, float *w, float *y, const int xdims[2], const int wdims[2], const int ydims[2]) {
  float *device_x, *device_w, *device_y;

  int size_x = sizeof(float) * xdims[0] * xdims[1];
  int size_w = sizeof(float) * wdims[0] * wdims[1];
  int size_y = sizeof(float) * ydims[0] * ydims[1];

  hipMalloc((void **)&device_x, size_x);
  hipMalloc((void **)&device_w, size_w);
  hipMalloc((void **)&device_y, size_y);

  hipMemcpy(device_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(device_w, w, size_w, hipMemcpyHostToDevice);

  fully_forward_kernel <<<xdims[0], xdims[1]>>> (device_x, device_w, device_y, xdims[0], xdims[1], wdims[1]);

  hipMemcpy(y, device_y, size_y, hipMemcpyDeviceToHost);

  hipFree(device_x);
  hipFree(device_w);
  hipFree(device_y);
}

// Choose the guess with largest score
static void argmax(const float *X, const int xdims[2], int *Y) {
	for (const auto i : range(0, xdims[0])) {
		auto max_idx = 0;
		auto max = X[i * xdims[1]];
		for (const auto j : range(0, xdims[1])) {
			const auto elem = X[(i * xdims[1]) + j];
			if (elem > max) {
				max_idx = j;
				max = elem;
			}
		}
		Y[i] = max_idx;
	}
}

// Forward operation for the CNN, a combination of conv layer + average pooling
// + relu
void forward_operation(float *x, float *conv1, float *conv2, float *fc1,
                       float *fc2, int *out) {
  // conv layer
  const int adims[] = {xdims[0], (xdims[1] - conv1dims[0] + 1),
                       (xdims[2] - conv1dims[1] + 1), conv1dims[3]};
  auto a = zeros<float>(adims);
  // conv_forward_valid(x, xdims, conv1, conv1dims, a, adims);

	const auto tic = now();

  conv_forward_unroll(x, conv1, a, xdims, conv1dims, adims);

	const auto toc = now();
	const auto elapsed = std::chrono::duration<double, std::milli>(toc - tic).count();;
	std::cout << "Calling f(args...) took " << elapsed << "milliseconds\n";

  // average pooling
  const int pool_size = 2;
  const int bdims[]   = {adims[0], adims[1] / pool_size, adims[2] / pool_size,
                         adims[3]};
  auto b = zeros<float>(bdims);

	const auto tic1 = now();

  // average_pool(a, adims, pool_size, b, bdims);
  average_pool_parallel(a, b, adims, bdims, pool_size);

	const auto toc1 = now();
	const auto elapsed1 = std::chrono::duration<double, std::milli>(toc1 - tic1).count();;
	std::cout << "Calling f(args...) took " << elapsed1 << "milliseconds\n";


  // conv layer
  const int cdims[] = {bdims[0], (bdims[1] - conv2dims[0] + 1),
                       (bdims[2] - conv2dims[1] + 1), conv2dims[3]};
  auto c = zeros<float>(cdims);

	const auto tic2 = now();

  // conv_forward_valid(b, bdims, conv2, conv2dims, c, cdims);
  conv_forward_unroll(b, conv2, c, bdims, conv2dims, cdims);

	const auto toc2 = now();
	const auto elapsed2 = std::chrono::duration<double, std::milli>(toc2 - tic2).count();;
	std::cout << "Calling f(args...) took " << elapsed2 << "milliseconds\n";

  // average pooling
  const int ddims[] = {cdims[0], cdims[1] / pool_size, cdims[2] / pool_size,
                       cdims[3]};
  auto d = zeros<float>(ddims);
  // average_pool(c, cdims, pool_size, d, ddims);

	const auto tic3 = now();

  average_pool_parallel(c, d, cdims, ddims, pool_size);

	const auto toc3 = now();
	const auto elapsed3 = std::chrono::duration<double, std::milli>(toc3 - tic3).count();;
	std::cout << "Calling f(args...) took " << elapsed3 << "milliseconds\n";

  // reshape
  const int ddims2[] = {ddims[0], ddims[1] * ddims[2] * ddims[3]};

  // matrix multiplication
  const int edims[] = {ddims[0], fc1dims[1]};
  auto e            = zeros<float>(edims);
  //fully_forward(d, ddims2, fc1, fc1dims, e, edims);
  fully_forward_parallel(d, fc1, e, ddims2, fc1dims, edims);

  // relu
  relu2(e, edims);

  // matrix multiplication
  const int fdims[] = {edims[0], fc2dims[1]};
  auto f            = zeros<float>(fdims);
  // fully_forward(e, edims, fc2, fc2dims, f, fdims);
  fully_forward_parallel(e, fc2, f, edims, fc2dims, fdims);

  argmax(f, fdims, out);

  delete[] a;
  delete[] b;
  delete[] c;
  delete[] d;
  delete[] e;
  delete[] f;
}

int main(int argc, char **argv) {

	if (argc != 3 && argc != 4) {
		std::cerr << "\n"
			<< "This program performs the forward opertion step for "
			"Convolutional Neural Network(CNN).  "
			"Sample usage: \n"
			<< argv[0]
			<< " [../data/test10.hdf5] [../data/model.hdf5] [10]\n";
		return -1;
	}
	FLAGS_testdata = std::string(argv[1]);
	FLAGS_model = std::string(argv[2]);
	if (argc == 3) {
		const std::map<std::string, int> default_batch_sizes{
			{ "../data/test2.hdf5", 2 },
			{ "../data/test10.hdf5", 10 },
			{ "../data/test100.hdf5", 100 },
			{ "../data/testfull.hdf5", 10000 } };
		const auto batch_size_in_map = default_batch_sizes.find(FLAGS_testdata);
		if (batch_size_in_map == default_batch_sizes.end()) {
			std::cerr << "\nERROR:: Unrecognized file " << FLAGS_testdata << " batch_size must be specified.\n";
			return -1;
		}
		FLAGS_batch_size = batch_size_in_map->second;
	}
	else if (argc == 4) {
		FLAGS_batch_size = atoi(argv[3]);
	}
	xdims[0] = FLAGS_batch_size;
	rdims[0] = FLAGS_batch_size;

	// Load data into x and y
	float *x = allocate<float>(xdims);
	float *y = allocate<float>(rdims);
	loadData(x, y);

	// Load model
	float *conv1 = allocate<float>(conv1dims);
	float *conv2 = allocate<float>(conv2dims);
	float *fc1 = allocate<float>(fc1dims);
	float *fc2 = allocate<float>(fc2dims);
	loadModel(conv1, conv2, fc1, fc2);

	// Perform foward opertion
	int *out = zeros<int>(FLAGS_batch_size);

	// get start time
	const auto start = now();

	forward_operation(x, conv1, conv2, fc1, fc2, out);

	// get end time
	const auto end = now();

	// get elapsed time in milliseconds
	const auto elapsed =
		std::chrono::duration<double, std::milli>(end - start).count();

	// Get reference
	int *ref = zeros<int>(FLAGS_batch_size);
	argmax(y, rdims, ref);

	// Calculate correctness
	int num_correct = 0;
	for (const auto i : range(0, FLAGS_batch_size)) {
		if (out[i] == ref[i]) {
			num_correct++;
		}
	}
	std::cout << "Done with " << FLAGS_batch_size << " queries in "
		<< "elapsed = " << elapsed << " milliseconds. Correctness: "
		<< static_cast<float>(num_correct) / FLAGS_batch_size << "\n";

	delete[] x;
	delete[] y;
	delete[] conv1;
	delete[] conv2;
	delete[] fc1;
	delete[] fc2;
	delete[] out;
	delete[] ref;

	return 0;
}
